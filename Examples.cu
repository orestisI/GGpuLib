#include "hip/hip_runtime.h"
#include "Examples.h"

void Examples::GpuArrayBasic(){
	GpuArrayOp<double> op(1024,1024);
	GpuArray<double> *a = new GpuArray<double>(2,3,4);
	GpuArray<double> *b = new GpuArray<double>(2,3,4);
	GpuArray<double> *c = new GpuArray<double>(2,3,1);
	GpuArray<double> *d = new GpuArray<double>(2,1,4);
	GpuArray<double> *e = new GpuArray<double>(2,3,4);
	op.Init(a,1.0);
	op.Init(b,1.0);
	op.LinearInit(c);
	op.LinearInit(d);
	op.LinearInit(e);

	GpuArray<double> *f = op.Add(c,1.0);
	GpuArray<double> *g = op.Add(d,1.0);
	GpuArray<double> *h = op.Add(e,1.0);
	GpuArray<double> *i = new GpuArray<double>(2,4,8);
	op.Init(i,1.0);
	
	std::cout<<"a = \n";
	a->Prnt();
	std::cout<<"b = \n";
	b->Prnt();
	std::cout<<"c = \n";
	c->Prnt();
	std::cout<<"d = \n";
	d->Prnt();
	std::cout<<"e = \n";
	e->Prnt();
	std::cout<<"f = \n";
	f->Prnt();
	std::cout<<"g = \n";
	g->Prnt();
	std::cout<<"h = \n";
	h->Prnt();
	std::cout<<"i = \n";
	i->Prnt();
	
	//Add
	GpuArray<double> *r1 = op.Add(a,b);
	std::cout<<"a + b = \n";
	r1->Prnt();
	//Add Bcast
	GpuArray<double> *r2 = op.Add(a,c);
	std::cout<<"a + c = \n";
	r2->Prnt();
	GpuArray<double> *r3 = op.Add(a,d);
	std::cout<<"a + d = \n";
	r3->Prnt();
	//Add Cnst
	GpuArray<double> *r4 = op.Add(a,2.0);
	std::cout<<"a + 2 = \n";
	r4->Prnt();

	//Sub
	GpuArray<double> *r5 = op.Sub(a,b);
	std::cout<<"a - b = \n";
	r5->Prnt();
	//Sub Bcast
	GpuArray<double> *r6 = op.Sub(a,c);
	std::cout<<"a - c = \n";
	r6->Prnt();
	GpuArray<double> *r7 = op.Sub(a,d);
	std::cout<<"a - d = \n";
	r7->Prnt();
	//Sub Cnst
	GpuArray<double> *r8 = op.Sub(a,1.0);
	std::cout<<"a - 1 = \n";
	r8->Prnt();

	//Dot
	GpuArray<double> *r9 = op.Dot(a,e);
	std::cout<<"a(dot)e = \n";
	r9->Prnt();
	//Dot Bcast
	GpuArray<double> *r10 = op.Dot(a,c);
	std::cout<<"a(dot)c = \n";
	r10->Prnt();
	GpuArray<double> *r11 = op.Dot(a,d);
	std::cout<<"a(dot)d = \n";
	r11->Prnt();
	//Dot Cnst
	GpuArray<double> *r12 = op.Dot(a,3.14);
	std::cout<<"a(dot)3.14 = \n";
	r12->Prnt();

	//Div
	GpuArray<double> *r13 = op.Div(a,h);
	std::cout<<"a/h = \n";
	r13->Prnt();
	//Div Bcast
	GpuArray<double> *r14 = op.Div(a,f);
	std::cout<<"a/f = \n";
	r14->Prnt();
	GpuArray<double> *r15 = op.Div(a,g);
	std::cout<<"a/g = \n";
	r15->Prnt();
	//Div Cnst
	GpuArray<double> *r16 = op.Div(a,2.0);
	std::cout<<"a/2 = \n";
	r16->Prnt();

	//Sigmoid
	GpuArray<double> *r17 = op.Sigmoid(e);
	std::cout<<"Sigmoid(e) = \n";
	r17->Prnt();

	//Log
	GpuArray<double> *r18 = op.Log(h);
	std::cout<<"Log(h) = \n";
	r18->Prnt();

	//Transpose
	GpuArray<double> *r19 = op.Transpose(e);
	std::cout<<"Transpose(e) = \n";
	r19->Prnt();
	
	GpuArray<double> *r20 = op.Mul(a,i);
	std::cout<<"a*i = \n";
	r20->Prnt();

	//Rnd
	GpuArray<double> *r21 = op.Rnd(2,5,6,-1.0,1.0);
	std::cout<<"Rnd = \n";
	r21->Prnt();

	//Add <-
	a->Add(e);
	std::cout<<"a = \n";
	a->Prnt();

	//Add Bcast <-
	std::cout<<"c = \n";
	c->Prnt();
	a->Add(c);
	std::cout<<"a = \n";
	a->Prnt();
	std::cout<<"d = \n";
	d->Prnt();
	a->Add(d);
	std::cout<<"a = \n";
	a->Prnt();

	//Add Cnst <-
	a->Add(3.14);
	std::cout<<"a = \n";
	a->Prnt();

	//Sub <-
	std::cout<<"e = \n";
	e->Prnt();
	std::cout<<"a = a - e\n";
	a->Sub(e);
	std::cout<<"a = \n";
	a->Prnt();

	//Sub Bcast <-
	std::cout<<"c = \n";
	c->Prnt();
	std::cout<<"a = a - c\n";
	a->Sub(c);
	std::cout<<"a = \n";
	a->Prnt();
	std::cout<<"d = \n";
	d->Prnt();
	std::cout<<"a = a - d\n";
	a->Sub(d);
	std::cout<<"a = \n";
	a->Prnt();

	//Sub Cnst <-
	std::cout<<"a = a - 1\n";
	a->Sub(1.0);
	std::cout<<"a = \n";
	a->Prnt();

	//Dot <-
	std::cout<<"a = a(dot)a\n";
	a->Dot(a);
	std::cout<<"a = \n";
	a->Prnt();

	//Dot Bcast <-
	std::cout<<"c = \n";
	c->Prnt();
	std::cout<<"a = a(dot)c\n";
	a->Dot(c);
	std::cout<<"a = \n";
	a->Prnt();
	std::cout<<"d = \n";
	d->Prnt();
	std::cout<<"a = a(dot)d\n";
	a->Dot(d);
	std::cout<<"a = \n";
	a->Prnt();
	
	//Dot Cnst <-
	std::cout<<"a = a(dot)2\n";
	a->Dot(2.0);
	std::cout<<"a = \n";
	a->Prnt();

	//Div <-
	std::cout<<"h = \n";
	h->Prnt();
	std::cout<<"a = a/h\n";
	a->Div(h);
	std::cout<<"a = \n";
	a->Prnt();

	//Div Bcast <-
	std::cout<<"f = \n";
	f->Prnt();
	std::cout<<"a = a/f\n";
	a->Div(f);
	std::cout<<"a = \n";
	a->Prnt();
	std::cout<<"g = \n";
	g->Prnt();
	std::cout<<"a = a/g\n";
	a->Div(g);
	std::cout<<"a = \n";
	a->Prnt();

	//Dot Cnst <-
	std::cout<<"a = a/2\n";
	a->Div(2.0);
	std::cout<<"a = \n";
	a->Prnt();

	//Sigmoid <-
	std::cout<<"h = \n";
	h->Prnt();
	std::cout<<"h = Sigmoid(h)\n";
	h->Sigmoid();
	std::cout<<"h = \n";
	h->Prnt();

	//Log <-
	std::cout<<"h = \n";
	h->Prnt();
	std::cout<<"h = Log(h)\n";
	h->Log();
	std::cout<<"h = \n";
	h->Prnt();

	delete a;
	delete b;
	delete c;
	delete d;
	delete e;
	delete f;
	delete g;
	delete h;
	delete r1;
	delete r2;
	delete r3;
	delete r4;
	delete r5;
	delete r6;
	delete r7;
	delete r8;
	delete r9;
	delete r10;
	delete r11;
	delete r12;
	delete r13;
	delete r14;
	delete r15;
	delete r16;
	delete r17;
	delete r18;
	delete r19;
	delete r20;
	delete r21;
}

void Examples::GpuArrayOpPower(){
	GpuArray<double> *a = new GpuArray<double>(2,3,4);
	GpuArrayOp<double> op(1024,1024);
	op.Init(a,2);
	a->Prnt();
	GpuArray<double> *b = op.Power(a,4);
	b->Prnt();

	delete a;
	delete b;
}

void Examples::GpuArrayOpPw(){
	GpuArray<double> *a = new GpuArray<double>(2,3,4);
	GpuArrayOp<double> op(1024,1024);

	op.Init(a,2);
	a->Prnt();
	GpuArray<double> *b = op.Pw(a,3);
	b->Prnt();

	delete a;
	delete b;
}

void Examples::GpuArrayOpMul3D(){
	GpuArrayOp<double> op(1024,1024);
	GpuArray<double> *a = op.Rnd(3,2,3,4,0,1.0);
	GpuArray<double> *b = op.Rnd(3,2,4,5,0,1.0);
	GpuArray<double> *c = op.Mul3D(a,b);
	a->Prnt();
	b->Prnt();
	c->Prnt();

	delete a;
	delete b;
	delete c;
}


void Examples::GpuArrayOpR(){
	GpuArrayOp<double> op(1024,1024);
	GpuArray<double> *input = op.Rnd(2,3,4,0,1.0);
	GpuArray<double> *Rx = op.Rx(input,0);
	GpuArray<double> *Ry = op.Ry(input,1);
	GpuArray<double> *Rz = op.Rz(input,2);

	input->Prnt();
	Rx->Prnt();
	Ry->Prnt();
	Rz->Prnt();

	delete input;
	delete Rx;
	delete Ry;
	delete Rz;
}

void Examples::GpuArrayOpV3D(){
	GpuArrayOp<double> op(1024,1024);
	GpuArray<double> *input = op.Rnd(2,3,4,0,1.0);
	GpuArray<double> *x = op.V3DX(input,0);
	GpuArray<double> *y = op.V3DY(input,1);
	GpuArray<double> *z = op.V3DZ(input,2);

	input->Prnt();
	x->Prnt();
	y->Prnt();
	z->Prnt();

	delete input;
	delete x;
	delete y;
	delete z;
}

void Examples::GpuArrayOpAddToRow(){
	GpuArrayOp<double> op(1024,1024);
	GpuArray<double> *a = op.Rnd(2,3,5,0,1.0);
	GpuArray<double> *a1 = op.AddToRow(a,0,0.1);
	GpuArray<double> *a2 = op.AddToRow(a,1,0.1);
	GpuArray<double> *a3 = op.AddToRow(a,2,0.1);

	a->Prnt();
	a1->Prnt();
	a2->Prnt();
	a3->Prnt();

	delete a;
	delete a1;
	delete a2;
	delete a3;
}

void Examples::GpuArrayOpTr3DZ(){
	GpuArrayOp<double> op(1024,1024);
	GpuArray<double> *a = op.Rnd(3,5,2,3,0,1.0);
	GpuArray<double> *b = op.Tr3DZ(a);
	a->Prnt();
	b->Prnt();

	delete a;
	delete b;
}

void Examples::GpuArrayOpTpu(){
	GpuArrayOp<double> op(1024,1024);
	GpuArray<double> *jac = op.Rnd(3,5,2,3,0,1.0);
	GpuArray<double> *var = op.Rnd(2,5,3,0,1.0);
	GpuArray<double> *tpu = op.Tpu(jac,var);

	jac->Prnt();
	var->Prnt();
	tpu->Prnt();

	delete jac;
	delete var;
	delete tpu;
}

void Examples::GpuArrayOp2DSt0(){
	GpuArrayOp<double> op(1024,1024);
	GpuArray<double> *a = op.Rnd(2,3,4,0,1.0);
	GpuArray<double> *b = op.Rnd(2,3,5,0,1.0);
	GpuArray<double> *c = op._2DSt0(a,b);

	a->Prnt();
	b->Prnt();
	c->Prnt();

	delete a;
	delete b;
	delete c;
}

void Examples::GpuArrayOp2DSt1(){
	GpuArrayOp<double> op(1024,1024);
	GpuArray<double> *a = op.Rnd(2,3,4,0,1.0);
	GpuArray<double> *b = op.Rnd(2,5,4,0,1.0);
	GpuArray<double> *c = op._2DSt1(a,b);

	a->Prnt();
	b->Prnt();
	c->Prnt();

	delete a;
	delete b;
	delete c;
}

void Examples::GpuArrayOp2DFourierSin(){
	GpuArrayOp<double> op(1024,1024);
	GpuArray<double> *a = op.Rnd(2,3,4,0,1.0);
	GpuArray<double> *fourierSin1 = op._2DFourierSin(a,1);
	GpuArray<double> *fourierSin2 = op._2DFourierSin(a,2);
	GpuArray<double> *fourierSin3 = op._2DFourierSin(a,3);

	a->Prnt();
	fourierSin1->Prnt();
	fourierSin2->Prnt();
	fourierSin3->Prnt();

	delete a;
	delete fourierSin1;
	delete fourierSin2;
	delete fourierSin3;
}

void Examples::GpuArrayOp2DFourierCos(){
	GpuArrayOp<double> op(1024,1024);
	GpuArray<double> *a = op.Rnd(2,3,4,0,1.0);
	GpuArray<double> *fourierCos1 = op._2DFourierCos(a,1);
	GpuArray<double> *fourierCos2 = op._2DFourierCos(a,2);
	GpuArray<double> *fourierCos3 = op._2DFourierCos(a,3);

	a->Prnt();
	fourierCos1->Prnt();
	fourierCos2->Prnt();
	fourierCos3->Prnt();

	delete a;
	delete fourierCos1;
	delete fourierCos2;
	delete fourierCos3;
}

void Examples::GpuArrayOp3DAlpha(){
	GpuArrayOp<double> op(1024,1024);
	GpuArray<double> *a = op.Rnd(3,4,3,6,0,1.0);
	GpuArray<double> *alpha = op._3DAlpha(a);

	a->Prnt();
	alpha->Prnt();

	delete a;
	delete alpha;
}

void Examples::GpuArrayOp3DPlaneL2(){
	GpuArrayOp<double> op(1024,1024);
	GpuArray<double> *a = op.Rnd(3,4,3,6,0,1.0);
	GpuArray<double> *plane = op._3DPlaneL2(a);

	a->Prnt();
	plane->Prnt();

	delete a;
	delete plane;
}

void Examples::GpuArrayPw(){
	GpuArray<double> *a = new GpuArray<double>(2,3,4);
	GpuArrayOp<double> op(1024,1024);

	op.Init(a,2);
	a->Pw(4);
	a->Prnt();

	delete a;
}

void Examples::GpuArraySet3D(){
	GpuArrayOp<double> op(1024,1024);
	GpuArray<double> *a = new GpuArray<double>(3,4,3,4);
	op.Init(a,0);
	a->Prnt();
	GpuArray<double> *a1 = op.Rnd(2,3,4,0,1.0);
	a1->Prnt();
	a->Set3D(a1,0);
	a->Prnt();
	GpuArray<double> *a2 = op.Rnd(2,3,4,0,1.0);
	a2->Prnt();
	a->Set3D(a2,1);
	a->Prnt();
	GpuArray<double> *a3 = op.Rnd(2,3,4,0,1.0);
	a3->Prnt();
	a->Set3D(a3,2);
	a->Prnt();
	GpuArray<double> *a4 = op.Rnd(2,3,4,0,1.0);
	a4->Prnt();
	a->Set3D(a4,3);
	a->Prnt();

	delete a;
	delete a1;
	delete a2;
	delete a3;
	delete a4;
}

void Examples::DNNSigmoidPrnt(){
	DNNSigmoid a(5,4,7,8,4,5,1024,1024);
	a.PrntWeights();
	a.PrntOffsets();
}

void Examples::DNNSigmoidFProp(){
	DNNSigmoid a(5,4,7,8,4,5,1024,1024);
	
	GpuArrayOp<double> op(1024,1024);
	GpuArray<double> *input = op.Rnd(2,4,10,-1.0,1.0);
	
	GpuArray<double> *res = a.FProp(input);
	res->Prnt();

	delete input;
	delete res;
	
}

void Examples::DNNSigmoidLoss(){
	DNNSigmoid a(5,4,7,8,4,2,1024,1024);
	GpuArrayOp<double> op(1024,1024);
	GpuArray<double> *input = op.Rnd(2,4,10,-1.0,1.0);
	GpuArray<double> *yHat = new GpuArray<double>(2,2,10);
	yHat->Set(0,0,0);
	yHat->Set(0,1,0);
	yHat->Set(0,2,1);
	yHat->Set(0,3,0);
	yHat->Set(0,4,0);
	yHat->Set(0,5,0);
	yHat->Set(0,6,1);
	yHat->Set(0,7,1);
	yHat->Set(0,8,1);
	yHat->Set(0,9,0);
	yHat->Set(1,0,0);
	yHat->Set(1,1,0);
	yHat->Set(1,2,1);
	yHat->Set(1,3,1);
	yHat->Set(1,4,1);
	yHat->Set(1,5,0);
	yHat->Set(1,6,1);
	yHat->Set(1,7,0);
	yHat->Set(1,8,1);
	yHat->Set(1,9,0);

	std::cout<<a.Loss(input,yHat)<<"\n";

	delete input;
	delete yHat;
}

void Examples::DNNSigmoidTrainBForse(){
	DNNSigmoid a(5,4,7,8,4,2,1024,1024);
	GpuArrayOp<double> op(1024,1024);
	GpuArray<double> *input = op.Rnd(2,4,10,-1.0,1.0);
	GpuArray<double> *yHat = new GpuArray<double>(2,2,10);
	yHat->Set(0,0,0);
	yHat->Set(0,1,0);
	yHat->Set(0,2,1);
	yHat->Set(0,3,0);
	yHat->Set(0,4,0);
	yHat->Set(0,5,0);
	yHat->Set(0,6,1);
	yHat->Set(0,7,1);
	yHat->Set(0,8,1);
	yHat->Set(0,9,0);
	yHat->Set(1,0,0);
	yHat->Set(1,1,0);
	yHat->Set(1,2,1);
	yHat->Set(1,3,1);
	yHat->Set(1,4,1);
	yHat->Set(1,5,0);
	yHat->Set(1,6,1);
	yHat->Set(1,7,0);
	yHat->Set(1,8,1);
	yHat->Set(1,9,0);

	a.TrainBForse(input,yHat,100);

	delete input;
	delete yHat;
}

void Examples::DNNSigmoidArrows(){//5013504
	int atributes = 5013504;
	int bachSize = 10;
	int bachNum = 10;
	DNNSigmoid a(3,atributes,10,1,1024,1024);
	GVector <GpuArray <double> > *input = new GVector<GpuArray<double> >(bachNum);
	GVector <GpuArray <double> > *yHat = new GVector<GpuArray<double> >(bachNum);
	CpuArray<double> *cpuInput,*cpuYHat;
	GpuArray<double> *gpuInput,*gpuYHat;
	GpuArrayOp<double> op(1024,1024);
	FILE *fp1,*fp2;

	fp1 = fopen("/home/orestis/JTC_Demo_CameraData/Train/Atributes","r");
	fp2 = fopen("/home/orestis/JTC_Demo_CameraData/Train/Labels","r");
	int val,label;
	for (int i=0; i<bachNum; i++){
		std::cout<<"Bach "<<i<<"\n";
		cpuInput = new CpuArray<double>(2,atributes,bachSize);
		cpuYHat = new CpuArray<double>(2,1,bachSize);
		for (int j=0; j<bachSize; j++){
			std::cout<<"Sample "<<j<<"\n";
			fscanf(fp2,"%d",&label);
			cpuYHat->Set(0,j,(double)label);
			for (int k=0; k<atributes; k++){
				fscanf(fp1,"%d",&val);
				cpuInput->Set(k,j,(double)val/255.0);
			}
		}
		gpuInput = op.Cpy(cpuInput);
		gpuYHat = op.Cpy(cpuYHat);
		delete cpuInput;
		delete cpuYHat;
		input->Set(i,gpuInput);
		yHat->Set(i,gpuYHat);
	}

	std::cout<<"Done Reading...\n";
	std::cout<<"Begining to Train...\n";

	a.TrainBProp(input,yHat,10);
	GpuArray<double> *out;
	
	for (int i=0; i<10; i++){
		out = a.FProp(input->Get(i));
		out->Prnt();
	}

	fclose(fp1);
	fclose(fp2);
	delete input;
	delete yHat;
}

void Examples::VFuncOut(){
	VFunc a(3,4,3,1024,1024);
	GpuArrayOp<double> op(1024,1024);
	GpuArray<double> *input = op.Rnd(2,3,10,-1.0,1.0);
	GpuArray<double> *out = a.Out(input);
	out->Prnt();

	delete input;
	delete out;
}

void Examples::VFuncTrain(){
	srand(time(NULL));
	VFunc a(1,6,1,1024,1024);
	GpuArrayOp<double> op(1024,1024);
	GpuArray<double> *input = op.Rnd(2,1,1000,-1.0,1.0);
	GpuArray<double> *f = op.Cos(input);
	a.Train(input,f,50000);
	GpuArray<double> *inputTest = op.Rnd(2,1,5,-1.0,1.0);
	inputTest->Prnt();
	GpuArray<double> *outTest = a.Out(inputTest);
	outTest->Prnt();
	a.PrntWeights();

	delete input;
	delete f;
	delete inputTest;
	delete outTest;
}

void Examples::_LexerNodeLeaf(){
	//
	//And Node
	//
	LexerNodeLeaf leafAnd;
	leafAnd.And("Orestis");
	leafAnd.TransitionMatrixPrnt();

	//
	//Or Node
	//
	LexerNodeLeaf leafOr;
	leafOr.Or("Orestis");
	leafOr.TransitionMatrixPrnt();
}

void Examples::_LexerToken(){
	LexerToken t(0,"Orestis",2,4);
	t.PrntData();
}

void Examples::_Lexer(){
	//Grammar: Orestis|Antwine
	//String: OrestisAntwineAntwineOrestis
	{
		LexerNodeRoot *root = new LexerNodeRoot();
		LexerNodeLeaf *orestis = new LexerNodeLeaf();
		orestis->And("Orestis");
		LexerNodeLeaf *antwine = new LexerNodeLeaf();
		antwine->And("Antwine");
		root->Add(orestis);
		root->Add(antwine);

		Lexer lexer(root);
		lexer.AddString("OrestisAntwineAntwineOrestis");
		LexerToken *t;
		t = lexer.GetNext();
		while(t->GetTokenNum() != -2){
			t->PrntToken();
			delete t;
			t = lexer.GetNext();
		}
	}
	printf("\n\n");
	//Grammar: {123456789}^{0123456789}*|Orestis
	//String: 1234Orestis
	{
		LexerNodeRoot *root = new LexerNodeRoot();
			
			LexerNodeLeaf *plus = new LexerNodeLeaf();
			plus->And("+");
			root->Add(plus);

			LexerNodeLeaf *minus = new LexerNodeLeaf();
			minus->And("-");
			root->Add(minus);

			LexerNodeLeaf *mul = new LexerNodeLeaf();
			mul->And("*");
			root->Add(mul);

			LexerNodeLeaf *div = new LexerNodeLeaf();
			div->And("/");
			root->Add(div);

			LexerNodeLeaf *digit =  new LexerNodeLeaf();
			digit->Or("0123456789");
			LexerNodeStar *digitStar = new LexerNodeStar(digit);
			root->Add(digitStar);

			LexerNodeLeaf *dNZdigit = new LexerNodeLeaf();
			dNZdigit->Or("123456789");
			LexerNodeLeaf *dDigit =  new LexerNodeLeaf();
			dDigit->Or("0123456789");
			LexerNodeStar *dDigitStar = new LexerNodeStar(dDigit);
			LexerNodeAnd *doubleFirstPart = new LexerNodeAnd();
			doubleFirstPart->Add(dNZdigit);
			doubleFirstPart->Add(dDigitStar);
			LexerNodeLeaf *dot = new LexerNodeLeaf();
			dot->And(".");
			LexerNodeLeaf *dDigit2 = new LexerNodeLeaf();
			dDigit2->Or("0123456789");
			LexerNodeStar *dDigitStar2 = new LexerNodeStar(dDigit2);
			LexerNodeAnd *dbl = new LexerNodeAnd();
			dbl->Add(doubleFirstPart);
			dbl->Add(dot);
			dbl->Add(dDigitStar2);
			root->Add(dbl);
	
			LexerNodeLeaf *dollar = new LexerNodeLeaf();
			dollar->And("$");
			LexerNodeLeaf *varDigit = new LexerNodeLeaf();
			varDigit->Or("0123456789");
			LexerNodeStar *varDigitStar = new LexerNodeStar(varDigit);
			LexerNodeAnd *var = new LexerNodeAnd();
			var->Add(dollar);
			var->Add(varDigitStar);
			root->Add(var);

			LexerNodeLeaf *rPar = new LexerNodeLeaf();
			rPar->And("(");
			root->Add(rPar);

			LexerNodeLeaf *lPar = new LexerNodeLeaf();
			lPar->And(")");
			root->Add(lPar);

			LexerNodeLeaf *tSin = new LexerNodeLeaf();
			tSin->And("sin");
			root->Add(tSin);

			LexerNodeLeaf *tCos = new LexerNodeLeaf();
			tCos->And("cos");
			root->Add(tCos);

			LexerNodeLeaf *tTan = new LexerNodeLeaf();
			tTan->And("tan");
			root->Add(tTan);

			LexerNodeLeaf *tArcSin = new LexerNodeLeaf();
			tArcSin->And("arcsin");
			root->Add(tArcSin);

			LexerNodeLeaf *tArcCos = new LexerNodeLeaf();
			tArcCos->And("arccos");
			root->Add(tArcCos);

			LexerNodeLeaf *tArcTan = new LexerNodeLeaf();
			tArcTan->And("arctan");
			root->Add(tArcTan);
			
			LexerNodeLeaf *tLog = new LexerNodeLeaf();
			tLog->And("log");
			root->Add(tLog);

			LexerNodeLeaf *tSqrt = new LexerNodeLeaf();
			tSqrt->And("sqrt");
			root->Add(tSqrt);

			LexerNodeLeaf *tExp = new LexerNodeLeaf();
			tExp->And("exp");
			root->Add(tExp);

			LexerNodeLeaf *tPow = new LexerNodeLeaf();
			tPow->And("^");
			root->Add(tPow);
			

		Lexer lexer(root);
		lexer.AddString("sin($2)*3+2*(2+cos($2*$3))");
		LexerToken *t;
		t = lexer.GetNext();
		while(t->GetTokenNum() != -2){
			t->PrntToken();
			delete t;
			t = lexer.GetNext();
		}
	}
}

void Examples::_PrntMatrixConnect(){
	PrntMatrix *prntMatrix = new PrntMatrix(100,100);
	prntMatrix->Init(' ');
	for (int i=0; i<10; i++){
		prntMatrix->Connect(rand()%100,rand()%100,rand()%100,rand()%100);
	}
	prntMatrix->Prnt();
	delete prntMatrix;
}

void Examples::_ASTPrnt(){

	ASTNode *a0 = new ASTNode();
	a0->SetId(0);
	a0->SetName("EXPR");
	
	ASTNode *a1 = new ASTNode();
	a1->SetId(1);
	a1->SetName("EXPR");

	ASTNode *a2 = new ASTNode();
	a2->SetId(2);
	a2->SetTerminal();
	a2->SetName("PLUS");

	ASTNode *a3 = new ASTNode();
	a3->SetId(3);
	a3->SetName("EXPR1");

	ASTNode *a4 = new ASTNode();
	a4->SetId(4);
	a4->SetName("EXPR1");

	ASTNode *a5 = new ASTNode();
	a5->SetId(5);
	a5->SetName("EXPR2");

	ASTNode *a6 = new ASTNode();
	a6->SetId(6);
	a6->SetName("EXPR1");

	ASTNode *a7 = new ASTNode();
	a7->SetId(7);
	a7->SetName("MUL");
	a7->SetTerminal();

	ASTNode *a8 = new ASTNode();
	a8->SetId(8);
	a8->SetName("EXPR2");

	ASTNode *a9 = new ASTNode();
	a9->SetId(9);
	a9->SetTerminal();
	a9->SetName("INT");

	ASTNode *a10 = new ASTNode();
	a10->SetId(10);
	a10->SetName("EXPR2");

	ASTNode *a11 = new ASTNode();
	a11->SetId(11);
	a11->SetName("INT");
	a11->SetTerminal();

	ASTNode *a12 = new ASTNode();
	a12->SetId(12);
	a12->SetName("INT");
	a12->SetTerminal();

	a10->Add(a12);
	a6->Add(a10);
	a8->Add(a11);
	a4->Add(a8);
	a4->Add(a7);
	a4->Add(a6);
	a1->Add(a4);
	a5->Add(a9);
	a3->Add(a5);
	a0->Add(a3);
	a0->Add(a2);
	a0->Add(a1);

	a0->Prnt();

	delete a0;
}

void Examples::_ASTGenFirstSet(){
	SingleProduction *S = new SingleProduction();
	S->Add(1);
	SingleProduction *Expr_0 = new SingleProduction();
	Expr_0->Add(1);
	Expr_0->Add(4);
	Expr_0->Add(2);
	SingleProduction *Expr_1 = new SingleProduction();
	Expr_1->Add(1);
	Expr_1->Add(5);
	Expr_1->Add(2);
	SingleProduction *Expr_2 = new SingleProduction();
	Expr_2->Add(2);
	SingleProduction *Expr1_0 = new SingleProduction();
	Expr1_0->Add(2);
	Expr1_0->Add(6);
	Expr1_0->Add(3);
	SingleProduction *Expr1_1 = new SingleProduction();
	Expr1_1->Add(2);
	Expr1_1->Add(7);
	Expr1_1->Add(3);
	SingleProduction *Expr1_2 = new SingleProduction();
	Expr1_2->Add(3);
	SingleProduction *Expr2_0 = new SingleProduction();
	Expr2_0->Add(8);
	SingleProduction *Expr2_1 = new SingleProduction();
	Expr2_1->Add(9);
	Expr2_1->Add(1);
	Expr2_1->Add(10);

	ASTGen astGen(11,4);
	astGen.Add(0,S);
	astGen.Add(1,Expr_0);
	astGen.Add(1,Expr_1);
	astGen.Add(1,Expr_2);
	astGen.Add(2,Expr1_0);
	astGen.Add(2,Expr1_1);
	astGen.Add(2,Expr1_2);
	astGen.Add(3,Expr2_0);
	astGen.Add(3,Expr2_1);

	astGen.ComputeFirstSet();
	astGen.PrntFirstSet();
}


void Examples::_ASTGenFollowSet(){
	SingleProduction *S = new SingleProduction();
	S->Add(1);
	SingleProduction *Expr_0 = new SingleProduction();
	Expr_0->Add(1);
	Expr_0->Add(4);
	Expr_0->Add(2);
	SingleProduction *Expr_1 = new SingleProduction();
	Expr_1->Add(1);
	Expr_1->Add(5);
	Expr_1->Add(2);
	SingleProduction *Expr_2 = new SingleProduction();
	Expr_2->Add(2);
	SingleProduction *Expr1_0 = new SingleProduction();
	Expr1_0->Add(2);
	Expr1_0->Add(6);
	Expr1_0->Add(3);
	SingleProduction *Expr1_1 = new SingleProduction();
	Expr1_1->Add(2);
	Expr1_1->Add(7);
	Expr1_1->Add(3);
	SingleProduction *Expr1_2 = new SingleProduction();
	Expr1_2->Add(3);
	SingleProduction *Expr2_0 = new SingleProduction();
	Expr2_0->Add(8);
	SingleProduction *Expr2_1 = new SingleProduction();
	Expr2_1->Add(9);
	Expr2_1->Add(1);
	Expr2_1->Add(10);

	ASTGen astGen(11,4);
	astGen.Add(0,S);
	astGen.Add(1,Expr_0);
	astGen.Add(1,Expr_1);
	astGen.Add(1,Expr_2);
	astGen.Add(2,Expr1_0);
	astGen.Add(2,Expr1_1);
	astGen.Add(2,Expr1_2);
	astGen.Add(3,Expr2_0);
	astGen.Add(3,Expr2_1);

	astGen.ComputeFirstSet();
	astGen.ComputeFollowSet();
	astGen.PrntFollowSet();
}

//S->Expr
//Expr->Expr + Expr1 | Expr - Expr1 | Expr1
//Expr1->Expr1 * Expr2 | Expr1 / Expr2 | Expr2
//Expr2->int | (Expr)
//S->0
//Expr->1
//Expr1->2
//Exzpr2->3
//+->4
//-->5
//*->6
///->7
//int->8
//(->9
//)->10
void Examples::_ASTGenStr(){
		LexerNodeRoot *root = new LexerNodeRoot();
		LexerNodeLeaf *plus = new LexerNodeLeaf();
		plus->And("+");
		root->Add(plus);
		LexerNodeLeaf *minus = new LexerNodeLeaf();
		minus->And("-");
		root->Add(minus);
		LexerNodeLeaf *mul = new LexerNodeLeaf();
		mul->And("*");
		root->Add(mul);
		LexerNodeLeaf *div = new LexerNodeLeaf();
		div->And("/");
		root->Add(div);
		LexerNodeLeaf *nonZeroDigit = new LexerNodeLeaf();
		nonZeroDigit->Or("123456789");
		LexerNodeLeaf *digit = new LexerNodeLeaf();
		digit->Or("0123456789");
		LexerNodeStar *digitStar = new LexerNodeStar(digit);
		LexerNodeAnd *integer = new LexerNodeAnd();
		integer->Add(nonZeroDigit);
		integer->Add(digitStar);
		root->Add(integer);
		LexerNodeLeaf *rPar = new LexerNodeLeaf();
		rPar->And("(");
		root->Add(rPar);
		LexerNodeLeaf *lPar = new LexerNodeLeaf();
		lPar->And(")");
		root->Add(lPar);

		Lexer *lexer = new Lexer(root);

		SingleProduction *S = new SingleProduction();
		S->Add(1);
		SingleProduction *Expr_0 = new SingleProduction();
		Expr_0->Add(1);
		Expr_0->Add(4);
		Expr_0->Add(2);
		SingleProduction *Expr_1 = new SingleProduction();
		Expr_1->Add(1);
		Expr_1->Add(5);
		Expr_1->Add(2);
		SingleProduction *Expr_2 = new SingleProduction();
		Expr_2->Add(2);
		SingleProduction *Expr1_0 = new SingleProduction();
		Expr1_0->Add(2);
		Expr1_0->Add(6);
		Expr1_0->Add(3);
		SingleProduction *Expr1_1 = new SingleProduction();
		Expr1_1->Add(2);
		Expr1_1->Add(7);
		Expr1_1->Add(3);
		SingleProduction *Expr1_2 = new SingleProduction();
		Expr1_2->Add(3);
		SingleProduction *Expr2_0 = new SingleProduction();
		Expr2_0->Add(8);
		SingleProduction *Expr2_1 = new SingleProduction();
		Expr2_1->Add(9);
		Expr2_1->Add(1);
		Expr2_1->Add(10);

		ASTGen astGen(11,4);
		astGen.Add(0,S);
		astGen.Add(1,Expr_0);
		astGen.Add(1,Expr_1);
		astGen.Add(1,Expr_2);
		astGen.Add(2,Expr1_0);
		astGen.Add(2,Expr1_1);
		astGen.Add(2,Expr1_2);
		astGen.Add(3,Expr2_0);
		astGen.Add(3,Expr2_1);

		astGen.SetName(0,"S");
		astGen.SetName(1,"EXPR");
		astGen.SetName(2,"EXPR1");
		astGen.SetName(3,"EXPR2");
		astGen.SetName(4,"+");
		astGen.SetName(5,"-");
		astGen.SetName(6,"*");
		astGen.SetName(7,"/");
		astGen.SetName(8,"INT");
		astGen.SetName(9,"(");
		astGen.SetName(10,")");

		astGen.SetLexer(lexer);
		astGen.ComputeFirstSet();
		astGen.ComputeFollowSet();
		astGen.ConstructRStruct();

		ASTNode *res = astGen.GetAST("18+2*(3+2)");
		if (res != NULL){
			res->Prnt();
		}
}

void Examples::_FOfSeVarPrntExpr(){
	FOfSeVar f(10);
	f.SetFun("5+2*sin($2*$3)+cos($2*(log($6)))");
	f.PrntExpr();
}

void Examples::_FOfSeVarPrntExprTree(){
	FOfSeVar f(10);
	f.SetFun("5+2*sin($2*$3)+cos($2*(log($6)))");
	f.PrntExprTree();
}

void Examples::_FOfSeVarGpuCodeGen(){
	FOfSeVar f(3);
	f.SetFun("5+2*sin($0*$2)+cos($1*(log($2)))");
	f.GpuCodeGen("GpuCodeGen.cu","MyfirstGpuAutoGeneratedFucntion");
}

void Examples::_FOfSeVarPDer(){
	FOfSeVar f(3);
	f.SetFun("($0)");
	FOfSeVar *fDer = f.GetPDer(0);
	fDer->GpuCodeGen("GpuCodeGen.cu","MyfirstGpuAutoGeneratedFucntion");
	delete fDer;
}






















